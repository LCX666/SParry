
#include <hip/hip_runtime.h>
// local memory 大小有限制 localSize 大小有限 只能容纳 100 * 1024 个点的
// 但是问题不大 整这个如果图的点太多 我们直接认为是大图 启用图分割 还是可以对
// 普通 delta 
__global__ void delta_stepping(int* V, int* E, int* W, int* n, int* s, int* delta, int* dist, int* predist, int* nowIsNull, int* quickBreak){
	
	const int u0 = (const int)threadIdx.x;
	const int offset = (const int) blockDim.x;
	const int localSize = 100; // 用这个来调控开启的多少

	// nowIsNull 全局变量 用于标记当前轮的桶是否是空的 1就是非空
	// quickBreak 为1 就说明有桶有点

	int B[localSize]; // 最多支持1024 * localSize 个点 也够了吧？ u 线程的 B[i] 实际意义是 代表点 i * blockDimx.x + u 归属于哪个桶
	bool hadin[localSize]; // 同理表示某个点是否曾经在当前桶中呆过  主要用于重边
	bool vis[localSize]; // 同理 就是标记某个点是否距离发生过改变 predist 会更快
	int id = 0; // 当前处理的桶的编号
	int u = -1; // 实际参与的代表结点
	int pos = -1; // 每个线程代表的点中的第几个点

	//// 初始化 局部变量
	for(int i = 0; i < localSize; i++){
		B[i] = -1;
		hadin[i] = 0;
		vis[i] = 0;
	}

	if(u0 == (*s) % offset){
		pos = (*s) / offset;
		B[pos] = 0; // 将源点标记为在0号桶中。 
		vis[pos] = 1;
		hadin[pos] = 1;
	}

	__syncthreads();

	while(1){ // 死循环
		//// 遍历一遍 以确认所有桶均为空了

		u = u0;
		while(u < (*n) && (*quickBreak) == 0){
			if(B[u / offset] != -1){ // 这说明至少一个点还在桶中，不予退出
				atomicExch(quickBreak, 1);
				//printf(">id = %2d, u = %3d 还在 Bid = %2d\n", id, u, B[u / offset]);
				break;
			}
			u += offset;
		}

		__syncthreads();
		
		if(*quickBreak == 0){ // 0就是 所有点都空了，不存在有桶有点了  就跳出循环
			// printf("我退出了 u = %d， id = %d\n", u, id);
			break;
		}
		
		//// 以下才是一轮更新桶中做的事
		while((*nowIsNull)){

			//// 遍历每一个点判断是否在当前桶中
			u = u0;
			while(u < *n){
				
				pos = u / offset;

				if(B[pos] == id){ // 当前结点u它属于当前桶中
					B[pos] = -1; // 从当前桶中剔除
					hadin[pos] = 1; // 那么当前这个点曾经也出现过在本桶中

					//printf("-id = %2d, u = %3d 剔除 Bid = %2d\n", id, u, id);
					
					if(vis[pos]){ // 当前结点u的距离发生过变化
						vis[pos] = 0; // 标记当前结点是距离在使用之后未发生过变化了
						
						for(int j = V[u]; j < V[u + 1]; j++){ // 枚举当前结点的所有邻居结点
							if(W[j] <= (*delta)){ // 轻边
								//printf("id = %d, 结点 u = %d 有这些轻边连接点: %d\n", id, u, E[j]);
								atomicMin(&predist[E[j]], dist[u] + W[j]);
							}
						}
					}
				} 
				u += offset;
			}

			//// 如果在一开进while就设置为空的话 有可能跑得快的先进来设置为0了，跑得慢的一看是0，就不进来了，就错了
			*nowIsNull = 0; // 标记当前桶空的 
			__syncthreads();

			//// 检测 predist 判断是否有点在本轮被更新了 同时检测是否有新点落入当前桶中
			u = u0;
			while(u < (*n)){
				if(predist[u] < dist[u]){ // 如果当前结点 predist 更小了，说明其可以被更新了
					
					pos = u / offset; // 计算的u相对偏移

					dist[u] = predist[u]; // 更新其 dist
					B[pos] = dist[u] / (*delta); // 计算更新后应该属于哪个桶
					vis[pos] = 1; // 标记其距离是发生过变化了
					//printf("+id = %2d, u = %3d 加入 Bid = %2d\n", id, u, B[pos]);

					if(B[pos] == id){ // 如果更新后还是落入当前桶中 那么就说明当前桶中是还有有点的
						// 当前桶中还有点 标记当前桶不是空的
						*nowIsNull = 1;
					}
				}
				u += offset;
			}
			__syncthreads();
		}

		//// 开始处理重边
		u = u0;
		while(u < (*n)){
			pos = u / offset;
			if(hadin[pos]){ // 当前结点在本轮中进入过当前桶中 进入过当前桶其距离是一定发生过改变的  所以不要vis来判断
				hadin[pos] = 0; // 标记为没有进入过了 下一个桶可以继续用
				
				for(int j = V[u]; j < V[u + 1]; j++){
					if(W[j] > (*delta)){ // 重边
						//printf("id = %d, 结点 u = %d 有这些重边连接点: %d\n", id, u, E[j]);
						atomicMin(&predist[E[j]], dist[u] + W[j]);
					}
				}
			}
			u += offset;
		}
		__syncthreads();

		//// 检测 predist 判断是否有点在本轮被更新了 同时检测是否有新点落入当前桶中
		u = u0;
		while(u < (*n)){
			if(predist[u] < dist[u]){ // 如果当前结点 predist 更小了，说明其可以被更新了
				
				pos = u / offset; // 计算的u相对偏移

				dist[u] = predist[u]; // 更新其 dist
				B[pos] = dist[u] / (*delta); // 计算更新后应该属于哪个桶
				vis[pos] = 1; // 标记其距离是发生过变化了
				
				//printf("+id = %d, u = %d 加入 Bid = %d\n", id, u, B[pos]);					
				
				// 由重边产生的更新不可能再落入当前桶中
				// if(B[pos] == id){ // 如果更新后还是落入当前桶中 那么就说明当前桶中是还有有点的
				// 	// 当前桶中还有点
				// 	标记当前桶不是空的
				// }
			}
			u += offset;
		}

		id += 1; // 进入下一个桶
		*nowIsNull = 1; // 假设下一轮的桶中有点
		*quickBreak = 0; // 假设所有桶都空了
		__syncthreads();
	}	
}