
#include <hip/hip_runtime.h>
// 简单并行 的 dijkstra
__global__ void dijkstra(int* V, int* E, int* W, int* n, int* vis, int* dist, int* predist){
	const int u0 = threadIdx.z * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x; // 每个thread有自己的编号 
	const int offset = blockDim.x * blockDim.y * blockDim.z; // 一个 block 里面有多少的thread
    __shared__ int quickBreak[1];
    
    int u = -1;

	for(int i = 0; i < (*n); i++){

		quickBreak[0] = 0;
		
		u = u0;
		while(u < *n){ 
			if(vis[u] == 0){ 
				vis[u] = 1;
				for(int j = V[u]; j < V[u + 1]; j++){ // 枚举u的终点，j是E和W数组的下标 E[j]是这条边的终点 W[j]是这条边的边权。	
					atomicMin(&predist[E[j]], dist[u] + W[j]);
				}
			}
			u += offset;
		}
		__syncthreads(); 

		u = u0;
		while(u < *n){
			if(predist[u] < dist[u]){ 
				dist[u] = predist[u];
				vis[u] = 0; 
				quickBreak[0] = 1;
			}
			u += offset;
		}
		__syncthreads(); 
		if(quickBreak[0] == 0)
			break;
	}	
}


/*下面这个是不使用多流的 使用默认流进行分块*/
// noStream 和下面的函数是一样的 就不再写了

/* 下面这个是 divide 也就是不使用多流的 使用默认流的*/
// 现在这个base是E中的起点 
__global__ void divide(int* V, int* E, int* W, int* n, int* flag, int* base, int* part, int* vis, int* dist, int* predist){
	
	const int u0 = threadIdx.z * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x; // 每个thread有自己的编号 
	const int offset = blockDim.x * blockDim.y * blockDim.z; // 一个 block 里面有多少的thread
	
	int u = -1;
	int l = -1;
	int r = -1;
	int localBase = base[0];
	int localPart = part[0];
	
	u = u0; // 此时的 u 不再是真的结点编号 来看是不是会超出 part 的范围, u + l 是 真的结点编号
	while(u < (*n)){ // 当前线程所代表的点在显存中

		if(V[u + 1] <= localBase){ // 自己右边
			u += offset;
			continue; // 这个结点的边不在合法范围内
		}
		else if(V[u] >= localBase + localPart){ // 自己的左边
			u += offset;
			continue; // 这个结点的边不在合法范围内
		}

		// 在上一轮更新过
		if(vis[u]){ 
			// 这个地方就不好判断了 因为分成的多块和多流之间的无先后顺序 故 vis 无法再使用
			//vis[u] -= 1; // 标记其松驰能力减一
			atomicSub(&vis[u], 1);

			// 对区间进行缩减
			l = localBase>V[u]?localBase:V[u];
			r = (localBase + localPart)<V[u + 1]?(localBase + localPart):V[u + 1];
			
			for(int j = l; j < r; j++){ // 枚举u的终点，j是E和W数组的下标 E[j]是这条边的终点 W[j]是这条边的边权。	
				atomicMin(&predist[E[j - localBase]], dist[u] + W[j - localBase]); // 注意原始的下标在现在的部分数组中是不对的 因此得映射一下
			}
		}
		
		u += offset;
	}
	__syncthreads(); 

	u = u0;
	while(u < (*n)){
		if(predist[u] < dist[u]){ 
			dist[u] = predist[u];
			vis[u] = (V[u + 1] + localPart - 1) / localPart - V[u] / localPart; // 重新计算其更新能力 
			flag[0] = 1;
		}
		u += offset;
	}
}
