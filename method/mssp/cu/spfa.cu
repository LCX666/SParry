
#include <hip/hip_runtime.h>
__global__ void kernelForMSSP(int *V, int *E, int *W, int *n, int *src, int *sn, bool *visit, int *dist, int *predist){
    /*thread 和 block 的两级复用 ，由于是不定多源，所以不能使用dist矩阵对称特性的利用，也不开启随机计算优化*/
    int u=0, stInd=0, st=0, align=0, old=0;
    __shared__ int QuickExit;
    const int blockId  = blockIdx.z *(gridDim.x *  gridDim.y) + blockIdx.y * gridDim.x + blockIdx.x;
    const int threadId = threadIdx.z*(blockDim.x * blockDim.y)+ threadIdx.y* blockDim.x+ threadIdx.x;
    const int blockSize= blockDim.x * blockDim.y * blockDim.z;
    const int gridSize = gridDim.x  * gridDim.y  *  gridDim.z;
    if(blockId >= (*sn)) return ;
    if(threadId >= (*n)) return ;
    stInd = blockId;
    st = src[stInd];
    while(st < (*n))
    {
        align = (stInd * (*n));
        while(1){/*这个while里解决了一个单元最短路问题*/
            QuickExit = 0;
            u = threadId;
            while(u < (*n)){
                if(visit[u + align]){
                    visit[u + align]=0;
                    for(int adj = V[u]; adj<V[u+1]; adj++){
                        old=atomicMin( &predist[align + E[adj]] , dist[align + u] + W[adj]);
                    }
                }
                u+=blockSize;
            }
            __syncthreads();
            u=threadId;
            while(u < (*n)){
                if(predist[align + u] < dist[align + u]){
                    dist[align + u] = predist[align + u];
                    visit[align + u] = 1;
                    QuickExit = 1;
                }
                u+=blockSize;
            }
            __syncthreads();
            if(QuickExit==0){
                break;
            }
        }
        __syncthreads();
        // /*这里开始dist中间结果利用*/
        // u=threadId;
        // while(u < (*n)){
        //     int ualign = u * (*n);
        //     old=atomicMin(&dist[ualign + st],dist[align + u]);
        //     if(old > dist[ualign + st]){
        //         visit[ualign + st]=1;
        //     }
        //     u+=blockSize;
        // }
        st += gridSize;
    }
}