
#include <hip/hip_runtime.h>
__global__ void kernelForMSSP(int *V, int *E, int *W, int *n, int *src, int *sn, bool *visit, int *dist, int *predist){
    int u=0, stInd=0, st=0, align=0, old=0;
    __shared__ int QuickExit;
    const int blockId  = blockIdx.z *(gridDim.x *  gridDim.y) + blockIdx.y * gridDim.x + blockIdx.x;
    const int threadId = threadIdx.z*(blockDim.x * blockDim.y)+ threadIdx.y* blockDim.x+ threadIdx.x;
    const int blockSize= blockDim.x * blockDim.y * blockDim.z;
    const int gridSize = gridDim.x  * gridDim.y  *  gridDim.z;
    if(blockId >= (*sn)) return ;
    if(threadId >= (*n)) return ;
    stInd = blockId;
    st = src[stInd];
    while(stInd < (*sn))
    {
        align = (stInd * (*n));
        while(1){ /* this while can solve a sssp*/ 
            QuickExit = 0;
            u = threadId;
            while(u < (*n)){
                if(visit[u + align]){
                    visit[u + align]=0;
                    for(int adj = V[u]; adj<V[u+1]; adj++){
                        old=atomicMin( &predist[align + E[adj]] , dist[align + u] + W[adj]);
                    }
                }
                u+=blockSize;
            }
            __syncthreads();
            u=threadId;
            while(u < (*n)){
                if(predist[align + u] < dist[align + u]){
                    dist[align + u] = predist[align + u];
                    visit[align + u] = 1;
                    QuickExit = 1;
                }
                u+=blockSize;
            }
            __syncthreads();
            if(QuickExit==0){
                break;
            }
        }
        __syncthreads();
        stInd += gridSize;
    }
}