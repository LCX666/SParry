
#include <hip/hip_runtime.h>
// 普通 edge 一个线程代表着一个边
__global__ void edge(int* src, int* des, int* w, int *n, int* m, int* dist){ // 每个线程作为一条边 判断两个端点是否发生了改变 w可以不用每次都传吧
	
	const int e0 = threadIdx.x; // 每个thread有自己的编号 
	const int offset = blockDim.x; // 一个 block 里面有多少的thread
    const int blockNum = (const int)(gridDim.x); // block总数 也即是 一次最多解决多少和单源问题

    int e = -1;
	int sn = -1;
    int s = blockIdx.x; // s是源点的问题
    int old = -1;
    
    __shared__ int quickBreak[1]; // block 内部的退出标识
    
    while(s < (*n)){ // 源点也必须有效才行
        sn = (s * (*n)); // 计算出当前源点的偏移
        
        while(1){
            e = e0;
            quickBreak[0] = 0;

            __syncthreads();
            
            while(e < (*m)){
                
                if (dist[src[e] + sn] > dist[des[e] + sn] + w[e]){
                    old = atomicMin(&dist[src[e] + sn], dist[des[e] + sn] + w[e]);

                    if(dist[src[e] + sn] < old){
                        quickBreak[0] = 1;
                    }
                }
                else if(dist[des[e] + sn] > dist[src[e] + sn] + w[e]){
                    old = atomicMin(&dist[des[e] + sn], dist[src[e] + sn] + w[e]);

                    if(dist[des[e] + sn] < old){
                        quickBreak[0] = 1;
                    }
                }
                e += offset;
            }
            
            __syncthreads();

            if(quickBreak[0] == 0){
                break;
            }
        }
        s += blockNum;
    }
}
