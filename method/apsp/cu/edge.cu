
#include <hip/hip_runtime.h>
// 普通 edge 一个线程代表着一个边
__global__ void edge(int* src, int* des, int* w, int *n, int* m, int* dist){ // 每个线程作为一条边 判断两个端点是否发生了改变 w可以不用每次都传吧
	
	const int e0 = threadIdx.z * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x; // 每个thread有自己的编号 
	const int offset = blockDim.x * blockDim.y * blockDim.z; // 一个 block 里面有多少的thread
	const int blockNum = (const int) gridDim.x * gridDim.y; // block 的数量

    int e = -1;
	int sn = -1;
    int s = blockIdx.z *(gridDim.x *  gridDim.y) + blockIdx.y * gridDim.x + blockIdx.x;
    int old = -1;
    
    __shared__ int quickBreak[1]; // block 内部的退出标识
    
    while(s < (*n)){ // 源点也必须有效才行
        sn = (s * (*n)); // 计算出当前源点的偏移
        
        while(1){
            e = e0;
            quickBreak[0] = 0;

            __syncthreads();
            
            while(e < (*m)){
            
                // if (dist[src[e] + sn] > dist[des[e] + sn] + w[e]){
                //     old = atomicMin(&dist[src[e] + sn], dist[des[e] + sn] + w[e]);

                //     if(dist[src[e] + sn] < old){
                //         quickBreak[0] = 1;
                //     }
                // }
                
                if(dist[des[e] + sn] > dist[src[e] + sn] + w[e]){
                    old = atomicMin(&dist[des[e] + sn], dist[src[e] + sn] + w[e]);

                    if(dist[des[e] + sn] < old){
                        quickBreak[0] = 1;
                    }
                }
                e += offset;
            }
            
            __syncthreads();

            if(quickBreak[0] == 0){
                break;
            }
        }
        s += blockNum;
    }
}
