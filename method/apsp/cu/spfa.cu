
#include <hip/hip_runtime.h>
__global__ void kernelForAPSP(int *V, int *E, int *W, int *n, bool *visit, int *dist, int *predist){
    
    const int blockId  = blockIdx.z *(gridDim.x *  gridDim.y) + blockIdx.y * gridDim.x + blockIdx.x;
    const int threadId = threadIdx.z*(blockDim.x * blockDim.y)+ threadIdx.y* blockDim.x+ threadIdx.x;
    const int blockSize =blockDim.x * blockDim.y * blockDim.z;
    const int gridSize = gridDim.x  * gridDim.y  * gridDim.z;
    __shared__ int QuickExit;
    int u=0, st=0, align=0, old=0;
    st = blockId;
    while(st < (*n))
    {
        align = (st * (*n));
        for(int rnd=0;rnd<(*n);rnd++){
            QuickExit = 0;
            u = threadId;
            while(u < (*n)){
                if(visit[u + align]){
                    visit[u + align]=0;
                    for(int adj = V[u];adj<V[u+1];adj++){
                        old=atomicMin( &predist[align + E[adj]] , dist[align + u] + W[adj]);
                    }
                }
                u+=blockSize;
            }
            __syncthreads();
            u=threadId;
            while(u < (*n)){
                if(predist[align + u] < dist[align + u]){
                    dist[align + u] = predist[align + u];
                    visit[align + u] = 1;
                    QuickExit = 1;
                }
                u+=blockSize;
            }
            __syncthreads();
            if(QuickExit==0){
                break;
            }
        }
        __syncthreads();
        st += gridSize;
    }
}