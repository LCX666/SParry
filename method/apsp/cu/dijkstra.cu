
#include <hip/hip_runtime.h>
// dijkstra 算法的并行自全源加未更新快速退出 

__global__ void dijkstra(int* V, int* E, int* W, int* n, int* vis, int* dist, int* predist){
	const int u0 = threadIdx.z * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x; // 每个thread有自己的编号 
	const int offset = blockDim.x * blockDim.y * blockDim.z; // 一个 block 里面有多少的thread
	const int blockNum = (const int) gridDim.x * gridDim.y; // block 的数量

	int u = -1;
	int sn = -1;
	int s = blockIdx.z *(gridDim.x *  gridDim.y) + blockIdx.y * gridDim.x + blockIdx.x;

	__shared__ int quickBreak[1];

	while(s < (*n)){ // 源点也必须有效才行

		sn = (s * (*n));

		for(int i = 0; i < (*n); i++){

			quickBreak[0] = 0; //应该是不需要原子操作，原子操作应该是更慢些。
			
			u = u0;
			while(u < *n){ 
				if(vis[u + sn] == 0){ 
					vis[u + sn] = 1;
					for(int j = V[u]; j < V[u + 1]; j++){ // 枚举u的终点，j是E和W数组的下标 E[j]是这条边的终点 W[j]是这条边的边权。	
						atomicMin(&predist[E[j] + sn], dist[u + sn] + W[j]); // s 为源点
					}
				}
				u += offset;
			}
			__syncthreads(); 

			u = u0;
			while(u < (*n)){
				if(predist[u + sn] < dist[u + sn]){ 
					dist[u + sn] = predist[u + sn];
					vis[u + sn] = 0; //后面再考虑把这个vis独立为自己的局部block变量 dist呢？

					quickBreak[0] = 1;
				}
				u += offset;
			}

			__syncthreads(); 
			if(quickBreak[0] == 0){
				break;
			}
		}
		s += blockNum; // 调向下一个源点 
	}	
}

/*下面这个是不使用多流的 使用默认流进行分块*/
// noStream 和下面的函数是一样的 就不再写了

/* 下面这个是 divide 也就是不使用多流的 使用默认流的*/
// 现在这个base是E中的起点 
__global__ void divide(int* V, int* E, int* W, int* n, int* flag, int* base, int* part, int* vis, int* dist, int* predist){
	
	const int u0 = threadIdx.z * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x; // 每个thread有自己的编号 
	const int offset = blockDim.x * blockDim.y * blockDim.z; // 一个 block 里面有多少的thread
	
	int u = -1;
	int l = -1;
	int r = -1;
	int localBase = base[0];
	int localPart = part[0];
	
	u = u0; // 此时的 u 不再是真的结点编号 来看是不是会超出 part 的范围, u + l 是 真的结点编号
	while(u < (*n)){ // 当前线程所代表的点在显存中

		if(V[u + 1] <= localBase){ // 自己右边
			u += offset;
			continue; // 这个结点的边不在合法范围内
		}
		else if(V[u] >= localBase + localPart){ // 自己的左边
			u += offset;
			continue; // 这个结点的边不在合法范围内
		}

		// 在上一轮更新过
		if(vis[u]){ 
			// 这个地方就不好判断了 因为分成的多块和多流之间的无先后顺序 故 vis 无法再使用
			//vis[u] -= 1; // 标记其松驰能力减一
			atomicSub(&vis[u], 1);

			// 对区间进行缩减
			l = localBase>V[u]?localBase:V[u];
			r = (localBase + localPart)<V[u + 1]?(localBase + localPart):V[u + 1];
			
			for(int j = l; j < r; j++){ // 枚举u的终点，j是E和W数组的下标 E[j]是这条边的终点 W[j]是这条边的边权。	
				atomicMin(&predist[E[j - localBase]], dist[u] + W[j - localBase]); // 注意原始的下标在现在的部分数组中是不对的 因此得映射一下
			}
		}
		
		u += offset;
	}
	__syncthreads(); 

	u = u0;
	while(u < (*n)){
		if(predist[u] < dist[u]){ 
			dist[u] = predist[u];
			vis[u] = (V[u + 1] + localPart - 1) / localPart - V[u] / localPart; // 重新计算其更新能力 
			flag[0] = 1;
		}
		u += offset;
	}
}
