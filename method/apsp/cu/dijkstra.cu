
#include <hip/hip_runtime.h>
// dijkstra 算法的并行自全源加未更新快速退出 
__global__ void dijkstra(int* V, int* E, int* W, int* n, int* vis, int* dist, int* predist){
	const int u0 = threadIdx.z * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x; 
	const int offset = blockDim.x * blockDim.y * blockDim.z; // the number of threads in a block
	const int blockNum = (const int) gridDim.x * gridDim.y; // the number of block

	int u = -1;
	int sn = -1;
	int s = blockIdx.z * (gridDim.x *  gridDim.y) + blockIdx.y * gridDim.x + blockIdx.x;

	__shared__ int quickBreak[1];

	while(s < (*n)){ // source must valid

		sn = (s * (*n));

		for(int i = 0; i < (*n); i++){

			quickBreak[0] = 0; 
			
			u = u0;
			while(u < *n){ 
				if(vis[u + sn] == 0){ 
					vis[u + sn] = 1;
					for(int j = V[u]; j < V[u + 1]; j++){ // the end of j, j is the index of E and W, E[j] is the end of this edge, W[j] is the weight of this edge.	
						atomicMin(&predist[E[j] + sn], dist[u + sn] + W[j]); // s is source.
					}
				}
				u += offset;
			}
			
			__syncthreads(); 

			u = u0;
			while(u < (*n)){
				if(predist[u + sn] < dist[u + sn]){ 
					dist[u + sn] = predist[u + sn];
					vis[u + sn] = 0; 

					quickBreak[0] = 1;
				}
				u += offset;
			}

			__syncthreads();

			if(quickBreak[0] == 0){
				break;
			}
			__syncthreads(); 
		}
		s += blockNum; // next vertex
	}	
}


// base is the start index of E  
__global__ void divide(int* V, int* E, int* W, int* n, int* flag, int* base, int* part, int* vis, int* dist, int* predist){
	
	const int u0 = threadIdx.z * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;  
	const int offset = blockDim.x * blockDim.y * blockDim.z; // the number of threads in a  block
	
	int u = -1;
	int l = -1;
	int r = -1;
	int localBase = base[0];
	int localPart = part[0];
	
	u = u0; // this turn, u is not the number of vertex, it's just a offset to tell out of range(part), and u+l is the true number of vertex.
	while(u < (*n)){ // the vertex represented by the current thread  is in video memory.

		if(V[u + 1] <= localBase){ // self right
			u += offset;
			continue; // the range of vertex's edge is illegal. 
		}
		else if(V[u] >= localBase + localPart){ // self left
			u += offset;
			continue; // the range of vertex's edge is illegal. 
		}

		// dist is updated
		if(vis[u]){ 
			// different part is not ordered, so can not use vis to tell.
			//vis[u] -= 1; // 
			atomicSub(&vis[u], 1); // set the ability of running sub one.

			// Shrink the range
			l = localBase>V[u]?localBase:V[u];
			r = (localBase + localPart)<V[u + 1]?(localBase + localPart):V[u + 1];
			
			for(int j = l; j < r; j++){ // get the end of u, j is the index of E and W, E[j] is the end of this edge, W[j] is the weight of this edge.	
				atomicMin(&predist[E[j - localBase]], dist[u] + W[j - localBase]); // the index is not true, need to add offset.
			}
		}
		
		u += offset;
	}
	__syncthreads(); 

	u = u0;
	while(u < (*n)){
		if(predist[u] < dist[u]){ 
			dist[u] = predist[u];
			vis[u] = (V[u + 1] + localPart - 1) / localPart - V[u] / localPart; // recalc the ability of running.
			flag[0] = 1;
		}
		u += offset;
	}
}
